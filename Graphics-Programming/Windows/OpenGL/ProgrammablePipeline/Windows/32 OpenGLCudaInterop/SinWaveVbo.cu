
#include <hip/hip_runtime.h>
__global__ void sinwave_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float animTime)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	float u = x / (float)width;
	float v = y / (float)height;
	
	u = (u*2.0)-1.0;
	v = (v*2.0)-1.0;
	
	float frequency = 4;
	float w = sinf(frequency*u+animTime) * cosf(frequency*v+animTime) * 0.5;
	
	pos[y*width+x] = make_float4(u,w,v,1.0);
	
	return;
}

void launchCudaKernel(float4* pos, unsigned int width, unsigned int height, float animTime)
{
	dim3 block(8, 8, 1);
	dim3 grid(width/block.x, height/block.y, 1.0);
	sinwave_vbo_kernel <<< grid, block >>> (pos, width, height, animTime);
}
